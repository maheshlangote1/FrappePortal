#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void mul_matrices(float *ad,float *bd,float *cd,int N)
{
        //cd[threadIdx.y * N + threadIdx.x] = ad[threadIdx.y * N + threadIdx.x] - bd[threadIdx.y * N + threadIdx.x];

        int pvalue=0;

        //find Row and Column corresponding to a data element for each thread
        int Row = blockIdx.y * blockDim.y + threadIdx.y;
        int Col = blockIdx.x * blockDim.x + threadIdx.x;

        //calculate dot product of Row of First Matrix and Column of Second Matrix
        for(int i=0;i< N;++i)
        {
                int m=ad[Row * N+i];
                int n=bd[i * N + Col];
                pvalue += m * n;
        }

        //store dot product at corresponding positon in resultant Matrix
        cd[Row * N + Col] = pvalue;

}
int main()
{
	int N = 4;				//size of square matrix
    unsigned int i,j;

	float *a,*b;
	float *ad,*bd,*cd,*c;
	size_t size=sizeof(float)* N * N;

    //allocate memory on host
	a=(float*)malloc(size);
	b=(float*)malloc(size);
    c=(float*)malloc(size);

    //allocate memory on device
	hipMalloc(&ad,size);
	//printf("\nAfter cudaMalloc for ad\n%s\n",cudaGetErrorString(cudaGetLastError()));
	hipMalloc(&bd,size);
	//printf("\nAfter cudaMalloc for bd\n%s\n",cudaGetErrorString(cudaGetLastError()));
    hipMalloc(&cd,size);
	//printf("\nAfter cudaMalloc for cd\n%s\n",cudaGetErrorString(cudaGetLastError()));

    //initialize host memory with its own indices
    for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
        {
			a[i * N + j]=(float)(i * N + j);
            b[i * N + j]= -(float)(i * N + j);
        }
    }

	//copy data from host memory to device memory
	hipMemcpy(ad,a,size,hipMemcpyHostToDevice);
    //printf("\nAfter HostToDevice Memcpy for ad\n%s\n",cudaGetErrorString(cudaGetLastError()));
	hipMemcpy(bd,b,size,hipMemcpyHostToDevice);
    //printf("\nAfter HostToDevice Memcpy for bd\n%s\n",cudaGetErrorString(cudaGetLastError()));

	//calculate execution configuration
    //dim3 blocksize (N, N);		//each block contains N * N threads, each thread calculates 1 data element

    //GPU timer code
    float time;
    hipEvent_t start,stop;
	hipEventCreate(&start);
    hipEventCreate(&stop);
	hipEventRecord(start,0);

    dim3 blocksize(2,2);		        //each block contains 16 * 16 (=256) threads
	dim3 gridsize(N/2,N/2);			//creating just sufficient no of blocks


    mul_matrices<<< gridsize, blocksize >>>(ad, bd, cd, N);

    hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time,start,stop);			//time taken in kernel call calculated
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipMemcpy(c,cd,size,hipMemcpyDeviceToHost);
	//printf("\nAfter DeviceToHost Memcpy for c \n%s\n",cudaGetErrorString(cudaGetLastError()));

	printf("Matrix A was---\n");
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
			printf("%f ",a[i*N+j]);
		printf("\n");
	}
	printf("\nMatrix B was---\n");
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
			printf("%f ",b[i*N+j]);
		printf("\n");
	}
	printf("\nAddition of A and B gives C----\n");
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
			printf("%f ",c[i*N+j]);              //if correctly evaluated, all values will be 0
		printf("\n");
	}

	printf("\n\nTime taken is %f (ms)\n",time);

    //deallocate host and device memories
    hipFree(ad); hipFree(bd); hipFree (cd);
	free(a);free(b);free(c);

	//_getch();
    return 1;
}
